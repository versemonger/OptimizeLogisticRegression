#include "hip/hip_runtime.h"
/*
 * All changes to code are copyright, 2017, Zhu Li, zhuli@unm.edu
 */

#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <iostream>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define THREAD_PER_BLOCK 512
#define SAMPLE_NUMBER (8*1024)
#define SAMPLE_ATTRIBUTE_NUMBER (32 * 32)
#define INITIAL_WEIGHTS_RANGE 0.01
#define SAMPLE_VALUE_RANGE 50
#define CONVERGE_RATE 0.0001
#define ITERATION_NUMBER 1000
#define MICROSEC_IN_SEC 1000000

//#define DEBUG
//#define WEIGHT_ZERO
/**
 *
 * @param n Length of the array.
 * @param range Range of the numbers in the array is [0, range].
 * @return An array filled with random numbers.
 */
float* generateRandomVectorFloat(int n, float range) {
  float* ptr = (float*)malloc(sizeof(float) * n);
  if (ptr != NULL) {
    for (int i = 0; i < n; i++) {
      ptr[i] =  (range * rand() / RAND_MAX) - range / 2;
    }
  }
  return ptr;
}

void output_device_vector(float* x, int length) {
  thrust::device_ptr<float> x_thr(x);
  thrust::device_vector<float> x_vector(x_thr, x_thr + length);
  thrust::copy(x_vector.begin(), x_vector.end(), std::ostream_iterator<float>(std::cout, "\t"));
  printf("\n");
}
/**
 *  return dot product of vector x and w.
 */
__host__ __device__ float dotProduct(float* x, float* w, int n) {
  float sum = 0;
  for (int i = 0; i < n; i++) {
    sum += x[i] * w[i];
  }
  return sum;
}


__host__ __device__ float logisticFunction(float* x, float* w, int n, float w0){
  float sum = w0 + dotProduct(x, w, n);
  return 1 / (1 + exp(sum));
}

__global__ void calculate_difference(float* delta, float* difference, float* x, float* weights, float w0, float* y) {
  __shared__ float shared_weights[SAMPLE_ATTRIBUTE_NUMBER];
  int tid = threadIdx.x;
  int i = blockDim.x * blockIdx.x + tid;
  if (tid < SAMPLE_ATTRIBUTE_NUMBER) {
    for (int j = 0; tid + j < SAMPLE_ATTRIBUTE_NUMBER; j += blockDim.x) {
      shared_weights[tid + j] = weights[tid + j];
    }
  }

  delta += i * SAMPLE_ATTRIBUTE_NUMBER;
  x += i * SAMPLE_ATTRIBUTE_NUMBER;
  __syncthreads();
  difference[i] = logisticFunction(x, shared_weights, SAMPLE_ATTRIBUTE_NUMBER, w0) + y[i] - 1;
  for (int j = 0; j < SAMPLE_ATTRIBUTE_NUMBER; j++) {
    *(delta + j) = *(x + j) * difference[i] * CONVERGE_RATE;
  }
}

__global__ void block_reduce(float *delta, float *weights, int block_number) {
  int j = blockDim.x * blockIdx.x + threadIdx.x;
  for (int i = 0; i < block_number; i++) {
    weights[j] += *(delta + i * SAMPLE_ATTRIBUTE_NUMBER + j);
  }
}

__global__ void reduce(float* delta, float* weights_grid) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int tid = threadIdx.x;
  int sum_holder_limit = blockDim.x / 2;
  int sum_stride = blockDim.x / 2;
  float* delta_temp = delta + i * SAMPLE_ATTRIBUTE_NUMBER;
  while (sum_stride > 0) {
    if (tid < sum_holder_limit) {
      for (int j = 0; j < SAMPLE_ATTRIBUTE_NUMBER; j++) {
        *(delta_temp + j) += *(delta_temp + sum_stride * SAMPLE_ATTRIBUTE_NUMBER + j);
      }
    }
    sum_holder_limit /= 2;
    sum_stride /= 2;
    __syncthreads();
  }
  if (tid == 0) {
    int weight_start = blockIdx.x * SAMPLE_ATTRIBUTE_NUMBER;
    int delta_start = blockDim.x * blockIdx.x * SAMPLE_ATTRIBUTE_NUMBER;
    for (int j = 0; j < SAMPLE_ATTRIBUTE_NUMBER; j++) {
      weights_grid[weight_start + j] += delta[delta_start + j];
    }
  }
}

int main() {

  srand(time(NULL));
  // initialize the weights randomly
  float w0 = (INITIAL_WEIGHTS_RANGE * rand() / RAND_MAX) - INITIAL_WEIGHTS_RANGE / 2;
  float* weights = generateRandomVectorFloat(SAMPLE_ATTRIBUTE_NUMBER, INITIAL_WEIGHTS_RANGE);
#ifdef WEIGHT_ZERO
  for (int i = 0; i < SAMPLE_ATTRIBUTE_NUMBER; i++) {
    weights[i] = 0;
  }
#endif
  float* x = (float*)malloc(SAMPLE_NUMBER * SAMPLE_ATTRIBUTE_NUMBER * sizeof(float));
  x = generateRandomVectorFloat(SAMPLE_NUMBER * SAMPLE_ATTRIBUTE_NUMBER, SAMPLE_VALUE_RANGE);

  // Set all benchmark weights as 0.5 or -0.5 randomly and generate the corresponding labels.
  // So we could test the effectiveness of the program according to whether
  // the program could predict the labels generated with benchmark weights
  float* y = (float*)malloc(SAMPLE_NUMBER * sizeof(float));
  float* benchMarkWeights = (float*)malloc(SAMPLE_ATTRIBUTE_NUMBER * sizeof(float));
  float benchMarkWeight0 = rand() % 2 - 0.5;
  for (int i = 0; i < SAMPLE_ATTRIBUTE_NUMBER; i++) {
    benchMarkWeights[i] = rand() % 2 - 0.5;
  }
  for (int i = 0; i < SAMPLE_NUMBER; i++) {
    y[i] = logisticFunction(x + i * SAMPLE_ATTRIBUTE_NUMBER, benchMarkWeights, SAMPLE_ATTRIBUTE_NUMBER, benchMarkWeight0) > 0.5 ? 0 : 1;
  }
  struct timeval tv;
  gettimeofday(&tv, NULL);
  long start = tv.tv_usec + tv.tv_sec * MICROSEC_IN_SEC;
  int block_number = SAMPLE_NUMBER / THREAD_PER_BLOCK;
  if (block_number == 0) {
    block_number = 1;
  }
  int thread_number = SAMPLE_NUMBER;
  if (thread_number > THREAD_PER_BLOCK) {
    thread_number = THREAD_PER_BLOCK;
  }

  int block_number_weights = SAMPLE_ATTRIBUTE_NUMBER / THREAD_PER_BLOCK;
  if (block_number_weights == 0) {
    block_number_weights = 1;
  }
  int thread_number_weights = SAMPLE_ATTRIBUTE_NUMBER;
  if (thread_number_weights > THREAD_PER_BLOCK) {
    thread_number_weights = THREAD_PER_BLOCK;
  }

  float *difference, *weight_device, *x_device, *y_device, *delta_device, *weight_grid;
  printf("Start memory alloc\t");
  gettimeofday(&tv, NULL);
  long diff = (tv.tv_sec * MICROSEC_IN_SEC + tv.tv_usec - start) / 1000;
  printf("Time taken: %ld seconds %ld milliseconds\n", diff / 1000, diff % 1000);
  hipMalloc((void**)&difference, SAMPLE_NUMBER * sizeof(float));
  hipMalloc((void**)&weight_device, SAMPLE_ATTRIBUTE_NUMBER * sizeof(float));
  hipMalloc((void**)&delta_device, SAMPLE_ATTRIBUTE_NUMBER * SAMPLE_NUMBER * sizeof(float));
  hipMalloc((void**)&x_device, SAMPLE_ATTRIBUTE_NUMBER * SAMPLE_NUMBER * sizeof(float));
  hipMalloc((void**)&y_device, SAMPLE_NUMBER * sizeof(float));
  hipMalloc((void**)&weight_grid, SAMPLE_ATTRIBUTE_NUMBER * block_number * sizeof(float));
  printf("Start memory copy\t");
  gettimeofday(&tv, NULL);
  diff = (tv.tv_sec * MICROSEC_IN_SEC + tv.tv_usec - start) / 1000;
  printf("Time taken: %ld seconds %ld milliseconds\n", diff / 1000, diff % 1000);
  hipMemcpy(x_device, x, SAMPLE_ATTRIBUTE_NUMBER * SAMPLE_NUMBER * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(y_device, y, SAMPLE_NUMBER * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(weight_device, weights, SAMPLE_ATTRIBUTE_NUMBER * sizeof(float), hipMemcpyHostToDevice);
#ifdef DEBUG
  printf("Block number:%d\n", block_number);
  printf("Thread number:%d\n", thread_number);
  printf("Original weights:\n");
  output_device_vector(weight_device, SAMPLE_ATTRIBUTE_NUMBER);
#endif
  printf("Start calculation\t");
  gettimeofday(&tv, NULL);
  long diff_start = (tv.tv_sec * MICROSEC_IN_SEC + tv.tv_usec - start) / 1000;
  printf("Time taken: %ld seconds %ld milliseconds\n", diff_start / 1000, diff_start % 1000);
  for (int k = 0; k < ITERATION_NUMBER; k++) {
    calculate_difference<<<block_number,thread_number>>>(delta_device, difference, x_device, weight_device, w0, y_device);
#ifdef DEBUG
    printf("x:\n");
    output_device_vector(x, SAMPLE_ATTRIBUTE_NUMBER * SAMPLE_NUMBER);
    printf("Delta:\n");
    output_device_vector(delta_device, SAMPLE_ATTRIBUTE_NUMBER * SAMPLE_NUMBER);
    printf("Difference:\n");
    output_device_vector(difference, SAMPLE_NUMBER);
    for (int i = 0; i < 10; i++) {
      printf("delta %d:\n", i);
      output_device_vector(delta_device + i * SAMPLE_ATTRIBUTE_NUMBER, SAMPLE_ATTRIBUTE_NUMBER);
    }
#endif
    reduce<<<block_number,thread_number>>>(delta_device, weight_grid);
    block_reduce<<<block_number_weights,thread_number_weights>>>(weight_grid, weight_device, block_number);
#ifdef DEBUG
    printf("weight_device after update:\n");
    output_device_vector(weight_device, SAMPLE_ATTRIBUTE_NUMBER);
#endif
  }
  printf("End calculation\t");
  gettimeofday(&tv, NULL);
  long diff_end = (tv.tv_sec * MICROSEC_IN_SEC + tv.tv_usec - start) / 1000;
  printf("Time taken: %ld seconds %ld milliseconds\n", diff_end / 1000, diff_end % 1000);
  float calculation_time = (diff_end - diff_start) * 1.0 / ITERATION_NUMBER;
  printf("Time taken by each kernel: %lf\n milliseconds",calculation_time);
  hipMemcpy(weights, weight_device, SAMPLE_ATTRIBUTE_NUMBER * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(x_device);
  hipFree(y_device);
  hipFree(weight_device);
  hipFree(difference);
#ifdef DEBUG
  for (int i = 0; i < SAMPLE_ATTRIBUTE_NUMBER; i++) {
    printf("Benchmark weight: %lf Estimated weight:%lf\n", benchMarkWeights[i], weights[i]);
  }
#endif
  // Predict the labels with weights estimated with logistic regression.
  float error = 0;
  for (int i = 0; i < SAMPLE_NUMBER; i++) {
    float predict = logisticFunction(x + i * SAMPLE_ATTRIBUTE_NUMBER, weights, SAMPLE_ATTRIBUTE_NUMBER, w0) > 0.5 ? 0 : 1;
#ifdef DEBUG
    printf("y[%d]: %lf Predicted: %lf\n", i, y[i], predict);
#endif
    error += fabs(predict - y[i]);
  }
  printf("Average error:%f\n", error / SAMPLE_NUMBER);
  printf("Finish verification\t");
  gettimeofday(&tv, NULL);
  diff = (tv.tv_sec * MICROSEC_IN_SEC + tv.tv_usec - start) / 1000;
  printf("Time taken: %ld seconds %ld milliseconds\n", diff / 1000, diff % 1000);
  free(x);
  free(y);
  free(weights);
  free(benchMarkWeights);
  return 0;
}



