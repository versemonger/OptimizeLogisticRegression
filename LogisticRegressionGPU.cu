#include "hip/hip_runtime.h"
/*
 * All changes to code are copyright, 2017, Zhu Li, zhuli@unm.edu
 */

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <iostream>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define THREAD_PER_BLOCK 512
#define SAMPLE_NUMBER (1024 * 8)
#define SAMPLE_ATTRIBUTE_NUMBER 32
#define INITIAL_WEIGHTS_RANGE 0.01
#define SAMPLE_VALUE_RANGE 50
#define CONVERGE_RATE 0.0001
#define ITERATION_NUMBER 10000
#define MICROSEC_IN_SEC 1000000

//#define DEBUG

/**
 *
 * @param n Length of the array.
 * @param range Range of the numbers in the array is [0, range].
 * @return An array filled with random numbers.
 */
float* generateRandomVectorFloat(int n, float range) {
  float* ptr = (float*)malloc(sizeof(float) * n);
  if (ptr != NULL) {
    for (int i = 0; i < n; i++) {
      ptr[i] =  (range * rand() / RAND_MAX) - range / 2;
    }
  }
  return ptr;
}

void output_device_vector(float* x, int length) {
  thrust::device_ptr<float> x_thr(x);
  thrust::device_vector<float> x_vector(x_thr, x_thr + length);
  thrust::copy(x_vector.begin(), x_vector.end(), std::ostream_iterator<float>(std::cout, "\t"));
  printf("\n");
}
/**
 *  return dot product of vector x and w.
 */
__host__ __device__ float dotProduct(float* x, float* w, int n) {
  float sum = 0;
  for (int i = 0; i < n; i++) {
    sum += x[i] * w[i];
  }
  return sum;
}


__host__ __device__ float logisticFunction(float* x, float* w, int n, float w0) {
  float sum = w0 + dotProduct(x, w, n);
  return 1 / (1 + exp(sum));
}

__global__ void calculate_difference(float* delta, float* difference, float* x, float* weights, float* w0, float* y) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  difference[i] = logisticFunction(x + i * SAMPLE_ATTRIBUTE_NUMBER, weights, SAMPLE_ATTRIBUTE_NUMBER, *w0) + y[i] - 1;
  int delta_index_start = i * SAMPLE_ATTRIBUTE_NUMBER;
  for (int j = 0; j < SAMPLE_ATTRIBUTE_NUMBER; j++) {
    //TODO: modify this after debug
    //*(delta + delta_index_start + j) = 0.0001 * i;
    *(delta + delta_index_start + j) = *(x + delta_index_start + j) * difference[i] * CONVERGE_RATE;
  }
}

__global__ void block_reduce(float *delta, float *weights, int block_number) {
  int j = threadIdx.x;
  for (int i = 0; i < block_number; i++) {
    weights[j] += *(delta + i * SAMPLE_ATTRIBUTE_NUMBER + j);
  }
}

__global__ void reduce(float* delta, float* weights_grid) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int tid = threadIdx.x;
  int sum_holder_limit = blockDim.x / 2;
  int sum_stride = blockDim.x / 2;
  float* delta_temp = delta + i * SAMPLE_ATTRIBUTE_NUMBER;
  while (sum_stride > 0) {
    if (tid < sum_holder_limit) {
      for (int j = 0; j < SAMPLE_ATTRIBUTE_NUMBER; j++) {
        *(delta_temp + j) += *(delta_temp + sum_stride * SAMPLE_ATTRIBUTE_NUMBER + j);
      }
    }
    sum_holder_limit /= 2;
    sum_stride /= 2;
    __syncthreads();
  }
  if (tid == 0) {
    int weight_start = blockIdx.x * SAMPLE_ATTRIBUTE_NUMBER;
    int delta_start = blockDim.x * blockIdx.x * SAMPLE_ATTRIBUTE_NUMBER;
    for (int j = 0; j < SAMPLE_ATTRIBUTE_NUMBER; j++) {
      weights_grid[weight_start + j] += delta[delta_start + j];
    }
  }
  __syncthreads();
}


__global__ void copy_weight(float* weight_device, float* new_weights) {
  for (int i = 0; i < SAMPLE_ATTRIBUTE_NUMBER; i++) {
    weight_device[i] = new_weights[i];
  }
}

struct offset_pointers {
    __device__ float* operator()(int offset, float* delta) {
      return offset + delta;
    }
};

struct sum_delta {
    __device__ float* operator() (float* x, float* y) {
      for (int i = 0; i < SAMPLE_ATTRIBUTE_NUMBER; i++) {
        *(x + i) += *(y + i);
      }
      return x;
    }
};



int main() {

  srand(time(NULL));
  // initialize the weights randomly
  float w0 = (INITIAL_WEIGHTS_RANGE * rand() / RAND_MAX) - INITIAL_WEIGHTS_RANGE / 2;
  float* weights = generateRandomVectorFloat(SAMPLE_ATTRIBUTE_NUMBER, INITIAL_WEIGHTS_RANGE);
  //TODO: delete this after debug
  for (int i = 0; i < SAMPLE_ATTRIBUTE_NUMBER; i++) {
    weights[i] = 0;
  }


  float* x = (float*)malloc(SAMPLE_NUMBER * SAMPLE_ATTRIBUTE_NUMBER * sizeof(float));
  x = generateRandomVectorFloat(SAMPLE_NUMBER * SAMPLE_ATTRIBUTE_NUMBER, SAMPLE_VALUE_RANGE);

  // Set all benchmark weights as 0.5 or -0.5 randomly and generate the corresponding labels.
  // So we could test the effectiveness of the program according to whether
  // the program could predict the labels generated with benchmark weights
  float* y = (float*)malloc(SAMPLE_NUMBER * sizeof(float));
  float* benchMarkWeights = (float*)malloc(SAMPLE_ATTRIBUTE_NUMBER * sizeof(float));
  float benchMarkWeight0 = rand() % 2 - 0.5;
  for (int i = 0; i < SAMPLE_ATTRIBUTE_NUMBER; i++) {
    benchMarkWeights[i] = rand() % 2 - 0.5;
  }
  for (int i = 0; i < SAMPLE_NUMBER; i++) {
    y[i] = logisticFunction(x + i * SAMPLE_ATTRIBUTE_NUMBER, benchMarkWeights, SAMPLE_ATTRIBUTE_NUMBER, benchMarkWeight0) > 0.5 ? 0 : 1;
  }
  struct timeval tv;
  gettimeofday(&tv, NULL);
  long start = tv.tv_usec + tv.tv_sec * MICROSEC_IN_SEC;
  int block_number = SAMPLE_NUMBER / THREAD_PER_BLOCK;
  if (block_number == 0) {
    block_number = 1;
  }
  int thread_number = SAMPLE_NUMBER;
  if (thread_number > THREAD_PER_BLOCK) {
    thread_number = THREAD_PER_BLOCK;
  }

  int block_number_weights = SAMPLE_ATTRIBUTE_NUMBER / THREAD_PER_BLOCK;
  if (block_number_weights == 0) {
    block_number_weights = 1;
  }
  int thread_number_weights = SAMPLE_ATTRIBUTE_NUMBER;
  if (thread_number_weights > THREAD_PER_BLOCK) {
    thread_number_weights = THREAD_PER_BLOCK;
  }

  float *difference, *weight_device, *x_device, *y_device, *w0_device, *delta_device, *weight_grid;// = (float *) malloc(sizeof(float) * SAMPLE_NUMBER);
  printf("Start memory alloc\n");
  hipMalloc((void**)&difference, SAMPLE_NUMBER * sizeof(float));
  hipMalloc((void**)&weight_device, SAMPLE_ATTRIBUTE_NUMBER * sizeof(float));
  hipMalloc((void**)&delta_device, SAMPLE_ATTRIBUTE_NUMBER * SAMPLE_NUMBER * sizeof(float));
  hipMalloc((void**)&x_device, SAMPLE_ATTRIBUTE_NUMBER * SAMPLE_NUMBER * sizeof(float));
  hipMalloc((void**)&y_device, SAMPLE_NUMBER * sizeof(float));
  hipMalloc((void**)&w0_device, sizeof(float));
  hipMalloc((void**)&weight_grid, SAMPLE_ATTRIBUTE_NUMBER * block_number * sizeof(float));
  printf("Start memory copy\n");
  hipMemcpy(x_device, x, SAMPLE_ATTRIBUTE_NUMBER * SAMPLE_NUMBER * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(w0_device, &w0, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(y_device, y, SAMPLE_NUMBER * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(weight_device, weights, SAMPLE_ATTRIBUTE_NUMBER * sizeof(float), hipMemcpyHostToDevice);
  thrust::device_vector<int> offset(SAMPLE_NUMBER);
  thrust::sequence(offset.begin(), offset.end(), 0, SAMPLE_ATTRIBUTE_NUMBER);
  //thrust::copy(offset.begin(), offset.end(), std::ostream_iterator<int>(std::cout, "\n"));
  thrust::device_vector<float*> delta_pointer(SAMPLE_NUMBER);
  thrust::fill(delta_pointer.begin(), delta_pointer.end(), delta_device);
  thrust::device_vector<float*> delta_pointers(SAMPLE_NUMBER);
  thrust::transform(offset.begin(), offset.end(), delta_pointer.begin(), delta_pointers.begin(), offset_pointers());
#ifdef DEBUG
  printf("Block number:%d\n", block_number);
  printf("Thread number:%d\n", thread_number);
  printf("Original weights:\n");
  output_device_vector(weight_device, SAMPLE_ATTRIBUTE_NUMBER);
#endif

  for (int k = 0; k < ITERATION_NUMBER; k++) {
    calculate_difference<<<block_number,thread_number>>>(delta_device, difference, x_device, weight_device, w0_device, y_device);
    hipDeviceSynchronize();
    reduce<<<block_number,thread_number>>>(delta_device, weight_grid);
    hipDeviceSynchronize();
    block_reduce<<<block_number_weights,thread_number_weights>>>(weight_grid, weight_device, block_number);
    hipDeviceSynchronize();
#ifdef DEBUG
    printf("x:\n");
    output_device_vector(x, SAMPLE_ATTRIBUTE_NUMBER * SAMPLE_NUMBER);
    printf("Delta:\n");
    output_device_vector(delta_device, SAMPLE_ATTRIBUTE_NUMBER * SAMPLE_NUMBER);
    printf("Difference:\n");
    output_device_vector(difference, SAMPLE_NUMBER);
    for (int i = 0; i < 10; i++) {
      printf("delta %d:\n", i);
      output_device_vector(delta_pointers[i], SAMPLE_ATTRIBUTE_NUMBER);
    }
//    for (int i = 2048; i < 2058; i++) {
//      printf("delta %d:\n", i);
//      output_device_vector(delta_pointers[i], SAMPLE_ATTRIBUTE_NUMBER);
//    }
//    for (int i = 4048; i < 4058; i++) {
//      printf("delta %d:\n", i);
//      output_device_vector(delta_pointers[i], SAMPLE_ATTRIBUTE_NUMBER);
//    }
//    for (int i = 6048; i < 6068; i++) {
//      printf("delta %d:\n", i);
//      output_device_vector(delta_pointers[i], SAMPLE_ATTRIBUTE_NUMBER);
//    }
    printf("weight_device after update:\n");
    output_device_vector(weight_device, SAMPLE_ATTRIBUTE_NUMBER);
#endif
//    float* newWeight = thrust::reduce(delta_pointers.begin(), delta_pointers.end(), weight_device, sum_delta());
//    hipDeviceSynchronize();
//#ifdef DEBUG
//    printf("New weights:\n");
//    output_device_vector(newWeight, SAMPLE_ATTRIBUTE_NUMBER);
//#endif
//    copy_weight<<<block_number_weights, thread_number_weights>>>(weight_device, newWeight);
//    hipDeviceSynchronize();
//#ifdef DEBUG

//    printf("\none cycle finished.\n\n\n");
//#endif
  }
  hipMemcpy(weights, weight_device, SAMPLE_ATTRIBUTE_NUMBER * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(x_device);
  hipFree(y_device);
  hipFree(weight_device);
  hipFree(difference);
  hipFree(w0_device);
#ifdef DEBUG
  for (int i = 0; i < SAMPLE_ATTRIBUTE_NUMBER; i++) {
    printf("Benchmark weight: %lf Estimated weight:%lf\n", benchMarkWeights[i], weights[i]);
  }
#endif
  // Predict the labels with weights estimated with logistic regression.
  float error = 0;
  for (int i = 0; i < SAMPLE_NUMBER; i++) {
    float predict = logisticFunction(x + i * SAMPLE_ATTRIBUTE_NUMBER, weights, SAMPLE_ATTRIBUTE_NUMBER, w0) > 0.5 ? 0 : 1;
#ifdef DEBUG
    printf("y[%d]: %lf Predicted: %lf\n", i, y[i], predict);
#endif
    error += fabs(predict - y[i]);
  }
  printf("Average error:%f\n", error / SAMPLE_NUMBER);
  gettimeofday(&tv, NULL);
  long diff = (tv.tv_sec * MICROSEC_IN_SEC + tv.tv_usec - start) / 1000;
  printf("Time taken: %ld seconds %ld milliseconds\n", diff / 1000, diff % 1000);
  free(x);
  free(y);
  free(weights);
  free(benchMarkWeights);
  return 0;
}



