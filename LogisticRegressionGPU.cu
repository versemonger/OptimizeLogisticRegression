#include "hip/hip_runtime.h"
/*
 * All changes to code are copyright, 2017, Zhu Li, zhuli@unm.edu
 */

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <iostream>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define SAMPLE_NUMBER 1024 * 4
#define SAMPLE_ATTRIBUTE_NUMBER 32 * 2
#define INITIAL_WEIGHTS_RANGE 0.01
#define SAMPLE_VALUE_RANGE 50
#define CONVERGE_RATE 0.0001
#define ITERATION_NUMBER 6000 * 2
#define MICROSEC_IN_SEC 1000000

//#define DEBUG

/**
 *
 * @param n Length of the array.
 * @param range Range of the numbers in the array is [0, range].
 * @return An array filled with random numbers.
 */
float* generateRandomVectorFloat(int n, float range) {
  float* ptr = (float*)malloc(sizeof(float) * n);
  if (ptr != NULL) {
    for (int i = 0; i < n; i++) {
      ptr[i] =  (range * rand() / RAND_MAX) - range / 2;
    }
  }
  return ptr;
}

/**
 *  return dot product of vector x and w.
 */
__host__ __device__ float dotProduct(float* x, float* w, int n) {
  float sum = 0;
  for (int i = 0; i < n; i++) {
    sum += x[i] * w[i];
  }
  return sum;
}


__host__ __device__ float logisticFunction(float* x, float* w, int n, float w0) {
  float sum = w0 + dotProduct(x, w, n);
  return 1 / (1 + exp(sum));
}


void updateWeights(float* weights, float** x, float* y, float w0, float* difference) {

  for (int i = 0; i < SAMPLE_NUMBER; i++) {
    difference[i] = logisticFunction(x[i], weights, SAMPLE_ATTRIBUTE_NUMBER, w0) + y[i] - 1;
  }

  for (int j = 0; j < SAMPLE_ATTRIBUTE_NUMBER; j++) {
    for (int i = 0; i < SAMPLE_NUMBER; i++) {
      weights[j] += x[i][j] * difference[i] * CONVERGE_RATE;
    }
  }
}


__global__ void calculate_difference(float* delta, float* difference, float* x, float* weights, float* w0, float* y) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  difference[i] = logisticFunction(x + i * SAMPLE_ATTRIBUTE_NUMBER, weights, SAMPLE_ATTRIBUTE_NUMBER, *w0) + y[i] - 1;
  int delta_index_start = i * SAMPLE_ATTRIBUTE_NUMBER;
  for (int j = 0; j < SAMPLE_ATTRIBUTE_NUMBER; j++) {
    *(delta + delta_index_start + j) = *(x + delta_index_start + j) * difference[i] * CONVERGE_RATE;
  }
}
__global__ void copy_weight(float* weight_device, float* new_weights) {
  for (int i = 0; i < SAMPLE_ATTRIBUTE_NUMBER; i++) {
    weight_device[i] = new_weights[i];
  }
}
__global__ void reduce(float* delta, float* weights) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int sum_holder_limit = SAMPLE_NUMBER / 2;
  int sum_stride = SAMPLE_NUMBER / 2;
  float* x = delta + i * SAMPLE_ATTRIBUTE_NUMBER;
  while (sum_stride > 0) {
    float* y = x + sum_stride * SAMPLE_ATTRIBUTE_NUMBER;
    if (i < sum_holder_limit) {

      for (int j = 0; j < SAMPLE_ATTRIBUTE_NUMBER; j++) {
        *(x + j) += *(y + j);
      }
    }
    sum_holder_limit /= 2;
    sum_stride /= 2;
    __syncthreads();
  }
  if (i == 0) {
    for (int j = 0; j < SAMPLE_ATTRIBUTE_NUMBER; j++) {
      weights[j] += delta[j];
    }
  }
}


struct offset_pointers {
    __device__ float* operator()(int offset, float* delta) {
      return offset + delta;
    }
};

struct sum_delta {
    __device__ float* operator() (float* x, float* y) {
      for (int i = 0; i < SAMPLE_ATTRIBUTE_NUMBER; i++) {
        *(x + i) += *(y + i);
      }
      return x;
    }
};

int main() {

  srand(time(NULL));
  // initialize the weights randomly
  float w0 = (INITIAL_WEIGHTS_RANGE * rand() / RAND_MAX) - INITIAL_WEIGHTS_RANGE / 2;
  float* weights = generateRandomVectorFloat(SAMPLE_ATTRIBUTE_NUMBER, INITIAL_WEIGHTS_RANGE);
  // TODO: load real data into x and y;
  // Generate random data for x

  float* x = (float*)malloc(SAMPLE_NUMBER * SAMPLE_ATTRIBUTE_NUMBER * sizeof(float));
  x = generateRandomVectorFloat(SAMPLE_NUMBER * SAMPLE_ATTRIBUTE_NUMBER, SAMPLE_VALUE_RANGE);

  // Set all benchmark weights as 0.5 or -0.5 randomly and generate the corresponding labels.
  // So we could test the effectiveness of the program according to whether
  // the program could predict the labels generated with benchmark weights
  float* y = (float*)malloc(SAMPLE_NUMBER * sizeof(float));
  float* benchMarkWeights = (float*)malloc(SAMPLE_ATTRIBUTE_NUMBER * sizeof(float));
  float benchMarkWeight0 = rand() % 2 - 0.5;
  for (int i = 0; i < SAMPLE_ATTRIBUTE_NUMBER; i++) {
    benchMarkWeights[i] = rand() % 2 - 0.5;
  }
  for (int i = 0; i < SAMPLE_NUMBER; i++) {
    y[i] = logisticFunction(x + i * SAMPLE_ATTRIBUTE_NUMBER, benchMarkWeights, SAMPLE_ATTRIBUTE_NUMBER, benchMarkWeight0) > 0.5 ? 0 : 1;
  }
  struct timeval tv;
  gettimeofday(&tv, NULL);
  long start = tv.tv_usec + tv.tv_sec * MICROSEC_IN_SEC;
  //clock_t start = clock(), diff;
  float *difference, *weight_device, *x_device, *y_device, *w0_device, *delta_device;// = (float *) malloc(sizeof(float) * SAMPLE_NUMBER);
  printf("Start memory alloc\n");
  hipMalloc((void**)&difference, SAMPLE_NUMBER * sizeof(float));
  hipMalloc((void**)&weight_device, SAMPLE_ATTRIBUTE_NUMBER * sizeof(float));
  hipMalloc((void**)&delta_device, SAMPLE_ATTRIBUTE_NUMBER * SAMPLE_NUMBER * sizeof(float));
  hipMalloc((void**)&x_device, SAMPLE_ATTRIBUTE_NUMBER * SAMPLE_NUMBER * sizeof(float));
  hipMalloc((void**)&y_device, SAMPLE_NUMBER * sizeof(float));
  hipMalloc((void**)&w0_device, sizeof(float));
  printf("Start memory copy\n");
  hipMemcpy(x_device, x, SAMPLE_ATTRIBUTE_NUMBER * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(w0_device, &w0, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(y_device, y, SAMPLE_NUMBER * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(weight_device, weights, SAMPLE_ATTRIBUTE_NUMBER * sizeof(float), hipMemcpyHostToDevice);
  thrust::device_vector<int> offset(SAMPLE_NUMBER);
  thrust::sequence(offset.begin(), offset.end(), 0, SAMPLE_ATTRIBUTE_NUMBER);
  thrust::copy(offset.begin(), offset.end(), std::ostream_iterator<int>(std::cout, "\n"));
  thrust::device_vector<float*> delta_pointer(SAMPLE_NUMBER);
  thrust::fill(delta_pointer.begin(), delta_pointer.end(), delta_device);
  thrust::device_vector<float*> delta_pointers(SAMPLE_NUMBER);
  thrust::transform(offset.begin(), offset.end(), delta_pointer.begin(), delta_pointers.begin(), offset_pointers());
  thrust::copy(delta_pointers.begin(), delta_pointers.end(), std::ostream_iterator<float*>(std::cout, "\n"));

  for (int k = 0; k < ITERATION_NUMBER; k++) {
    calculate_difference<<<16,64>>>(delta_device, difference, x_device, weight_device, w0_device, y_device);
    hipDeviceSynchronize();
     float* newWeight = thrust::reduce(delta_pointers.begin(), delta_pointers.end(), weight_device, sum_delta());
    hipDeviceSynchronize();
    copy_weight<<<1, 64>>>(weight_device, newWeight);
    hipDeviceSynchronize();
  }
  hipMemcpy(weights, weight_device, SAMPLE_ATTRIBUTE_NUMBER * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(x_device);
  hipFree(y_device);
  hipFree(weight_device);
  hipFree(difference);
  hipFree(w0_device);
#ifdef DEBUG
  for (int i = 0; i < SAMPLE_ATTRIBUTE_NUMBER; i++) {
    printf("Benchmark weight: %lf Estimated weight:%lf\n", benchMarkWeights[i], weights[i]);
  }
#endif
  // Predict the labels with weights estimated with logistic regression.
  float error = 0;
  for (int i = 0; i < SAMPLE_NUMBER; i++) {
    float predict = logisticFunction(x + i * SAMPLE_ATTRIBUTE_NUMBER, weights, SAMPLE_ATTRIBUTE_NUMBER, w0) > 0.5 ? 0 : 1;
#ifdef DEBUG
    printf("y[%d]: %lf Predicted: %lf\n", i, y[i], predict);
#endif
    error += fabs(predict - y[i]);
  }
  printf("Average error:%lf\n", error / SAMPLE_NUMBER);
  //int diff = gettimeofday() - start;
  //diff = clock() - start;
  gettimeofday(&tv, NULL);
  long diff = (tv.tv_sec * MICROSEC_IN_SEC + tv.tv_usec - start) / 1000;
  printf("Time taken: %ld seconds %ld milliseconds\n", diff / 1000, diff % 1000);
  free(x);
  free(y);
  free(weights);
  free(benchMarkWeights);
  return 0;
}



